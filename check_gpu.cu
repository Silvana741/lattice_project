#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;

int main() {
    int device_count = 0;
    hipError_t error = hipGetDeviceCount(&device_count);

    if (error != hipSuccess) {
        cout << "CUDA error: " << hipGetErrorString(error) << endl;
        return -1;
    }

    if (device_count == 0) {
        cout << "No CUDA-capable GPU detected!" << endl;
    } else {
        cout << "Number of CUDA-capable GPUs: " << device_count << endl;
    }

    return 0;
}
